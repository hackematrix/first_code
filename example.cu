
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void kernel() {
    printf("Hello from CUDA kernel!\n");
}

int main() {
    kernel<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}

